#include "helper.h"

void blas2_test() {

    hipModule_t cuModule;
    hipFunction_t cuFunction;
    char* ModuleFile2 = "blas2.ptx";
    char* KernelName2_0 = "blas2";

    const int N = 1024;
    const int THREADS_PER_BLOCK = 1024;
    const int NUMBER_OF_BLOCKS = 32;
    const int GRID_DIM_X = NUMBER_OF_BLOCKS;
    const int GRID_DIM_Y = 1;
    const int GRID_DIM_Z = 1;
    const int BLOCK_DIM_X = THREADS_PER_BLOCK;
    const int BLOCK_DIM_Y = 1;
    const int BLOCK_DIM_Z = 1;

    // Number of bytes to allocate
    int Nsize = N * sizeof(float);
    int MatSize = (N*M) * sizeof(float);

    // Load precompiled PTX from nvcc -> cuModule
    cudaChk(hipModuleLoad(&cuModule, ModuleFile));
        
    // Get function handle from module -> cuFunction
    cudaChk(hipModuleGetFunction(&cuFunction, cuModule, KernelName));

    // Allocate vectors in host memory
    float* a = (float*)malloc(size);
    float* b = (float*)malloc(size);
    float* c = (float*)malloc(size);

    // Allocate vectors in device memory
    hipDeviceptr_t dev_a, dev_b, dev_c;
    cudaChk(hipMalloc(&dev_a, size));
    cudaChk(hipMalloc(&dev_b, size));
    cudaChk(hipMalloc(&dev_c, size);

    // Initialize host vectors
    initArr(3, a, N);
    initArr(4, b, N);
    initArr(0, c, N);

    // Copy vectors from host memory to device memory
    cudaChk(hipMemcpyHtoD(dev_a, a, size));
    cudaChk(hipMemcpyHtoD(dev_b, b, size));

    // setup kernel arguments (using the simple kernel argument format)
    unsigned int sharedMemBytes = 1;
    hipStream_t hStream = 0;
    void* args[] = { &dev_a, &dev_b, &dev_c, M, N };

    // Launch the kernel on device
    TIMER_START
    cudaChk(hipModuleLaunchKernel(cuFunction, \
        GRID_DIM_X, GRID_DIM_Y, GRID_DIM_Z, \
        BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, \
        sharedMemBytes, hStream, args, 0));
    cudaChk(hipCtxSynchronize());
    TIMER_STOP

    // Check for errors
    hipError_t cuErrSync  = hipGetLastError();
    if (cuErrSync != hipSuccess) printf("sync error: %s\n", hipGetErrorString(cuErrSync));
    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess) printf("asyc error:  %s\n", hipGetErrorString(asyncErr0));

    // Retrieve results from device & verify/use
    cudaChk(hipMemcpyDtoH(c, dev_c, size));

    // Check data for correctness
    checkElementsAre(7, c, N);

    // Free Host Memory
    free( a ); 
    free( b ); 
    free( c );

    // Free Device Memory
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
}