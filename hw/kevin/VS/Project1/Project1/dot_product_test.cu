#include "hip/hip_runtime.h"
#include "helper.h"

void dot_product_test() {

    hipModule_t cuModule;
    hipFunction_t cuFunction;
    char* ModuleFile = "dot_product.ptx";
    char* KernelName0 = "dot_product_float";
    char* KernelName1 = "dot_product_float2";
    char* KernelName2 = "dot_product_float4";

    const int N = 1024;
    const int THREADS_PER_BLOCK = 1024;
    const int NUMBER_OF_BLOCKS = 32;
    const int GRID_DIM_X = NUMBER_OF_BLOCKS;
    const int GRID_DIM_Y = 1;
    const int GRID_DIM_Z = 1;
    const int BLOCK_DIM_X = THREADS_PER_BLOCK;
    const int BLOCK_DIM_Y = 1;
    const int BLOCK_DIM_Z = 1;

    // Number of bytes to allocate
    int size = N * sizeof(float); // float2, float4

    // Load precompiled PTX from nvcc -> cuModule
    cudaChk(hipModuleLoad(&cuModule, ModuleFile));
        
    // Get function handle from module -> cuFunction
    cudaChk(hipModuleGetFunction(&cuFunction, cuModule, KernelName));

    // Allocate vectors in host memory
    int P = 3;
    float* a = (float*)malloc(size);
    float* b = (float*)malloc(size);
    float* c = (float*)malloc(size/(P*sizeof(int)));

    // Allocate vectors in device memory
    hipDeviceptr_t dev_a, dev_b, dev_c;
    cudaChk(hipMalloc(&dev_a, size));
    cudaChk(hipMalloc(&dev_b, size));
    cudaChk(hipMalloc(&dev_c, size);

    // Initialize host vectors
    initArr(3, a, N);
    initArr(4, b, N);
    initArr(0, c, N/P);

    // Copy vectors from host memory to device memory
    cudaChk(hipMemcpyHtoD(dev_a, a, size));
    cudaChk(hipMemcpyHtoD(dev_b, b, size));

    // setup kernel arguments (using the simple kernel argument format)
    unsigned int sharedMemBytes = 1;
    hipStream_t hStream = 0;
    void* args[] = { &dev_a, &dev_b, &dev_c, M, P };

    // Launch the kernel on device
    TIMER_START
    cudaChk(hipModuleLaunchKernel(cuFunction, \
        GRID_DIM_X, GRID_DIM_Y, GRID_DIM_Z, \
        BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, \
        sharedMemBytes, hStream, args, 0));
    cudaChk(hipCtxSynchronize());
    TIMER_STOP

    // Check for errors
    hipError_t cuErrSync  = hipGetLastError();
    if (cuErrSync != hipSuccess) printf("sync error: %s\n", hipGetErrorString(cuErrSync));
    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess) printf("asyc error:  %s\n", hipGetErrorString(asyncErr0));

    // Retrieve results from device & verify/use
    cudaChk(hipMemcpyDtoH(c, dev_c, size));

    // Check data for correctness
    float fTol = 1e-7;
    if (abs(dotproduct(a,b)-sumArr(c))> ftol) {
        printf("FAIL: dot product kernel doesn't match CPU: %0.0f != %0.0f",dotproduct(a,b), sumArr(c));
    }

    // Free Host Memory
    free( a ); 
    free( b ); 
    free( c );

    // Free Device Memory
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
}
