﻿#include <Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "helper.h"

#define N 1024
#define M 1024

#define TIMER_INIT \
    LARGE_INTEGER frequency; \
    LARGE_INTEGER t1,t2; \
    double elapsedTime; \
    QueryPerformanceFrequency(&frequency);

#define TIMER_START QueryPerformanceCounter(&t1);

#define TIMER_STOP \
    QueryPerformanceCounter(&t2); \
    elapsedTime=(float)(t2.QuadPart-t1.QuadPart)/frequency.QuadPart; \
    printf("0.000000%f sec\n", elapsedTime);

void matrix_add_test() {

    TIMER_INIT
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    char* ModuleFile = (char*)"matrix_add.ptx";
    char* KernelName = (char*)"matrix_add";

    const int THREADS_PER_BLOCK = 1024;
    const int NUMBER_OF_BLOCKS = 32;
    const int GRID_DIM_X = 4;
    const int GRID_DIM_Y = 1;
    const int GRID_DIM_Z = 1;
    const int BLOCK_DIM_X = THREADS_PER_BLOCK;
    const int BLOCK_DIM_Y = 1;
    const int BLOCK_DIM_Z = 1;

    // Number of bytes to allocate for MxN matrix with type float
    int size = (M * N) * sizeof(float);

    // Load precompiled PTX from nvcc -> cuModule
    hipModuleLoad(&cuModule, ModuleFile);

    // Get function handle from module -> cuFunction
    hipModuleGetFunction(&cuFunction, cuModule, KernelName);

    // Allocate vectors in host memory
    float** a = (float**)malloc(size);
    float** b = (float**)malloc(size);
    float** c = (float**)malloc(size);

    // Allocate vectors in device memory
    hipDeviceptr_t dev_a, dev_b, dev_c;
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_c, size);

    // Initialize host vectors
    initMat(3, a, M, N);
    initMat(4, b, M, N);
    initMat(0, c, M, N);

    // Copy vectors from host memory to device memory
    hipMemcpyHtoD(dev_a, a, size);
    hipMemcpyHtoD(dev_b, b, size);

    // setup kernel arguments (using the simple kernel argument format)
    unsigned int sharedMemBytes = 1;
    hipStream_t hStream = 0;
    void* args[] = { &dev_a, &dev_b, &dev_c };

    // Launch the kernel on device
    TIMER_START
        hipModuleLaunchKernel(cuFunction, \
            GRID_DIM_X, GRID_DIM_Y, GRID_DIM_Z, \
            BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, \
            sharedMemBytes, hStream, args, 0);
    hipCtxSynchronize();
    TIMER_STOP

    // Check for errors
    hipError_t cuErrSync = hipGetLastError();
    if (cuErrSync != hipSuccess) printf("sync error: %s\n", hipGetErrorString(cuErrSync));
    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess) printf("asyc error:  %s\n", hipGetErrorString(cuErrAsync));

    // Retrieve results from device & verify/use
    hipMemcpyDtoH(c, dev_c, size);

    // Check data for correctness
    checkElementsMat(7, c, M, N);

    // Free Host Memory
    free(a);
    free(b);
    free(c);

    // Free Device Memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}