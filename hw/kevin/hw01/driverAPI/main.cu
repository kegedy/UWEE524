#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_add_test.cuh"
#include "helper.h"

#define cudaChk(err)  __cudaChk(err, __FILE__, __LINE__)

inline void __cudaChk(hipError_t err, const char* file, const int line)
{
    if (hipSuccess != err) {
        fprintf(stderr,
            "CUDA Driver API error = %04d from file <%s>, line %i.\n",
            err, file, line);
        exit(0);
    }
}

int main(int argc, char** argv) {
    
    // Define static variables
    int deviceCount = 0;
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
   
    // Initialize GPU host API 
    cudaChk(hipInit(0));
    
    // Query for device information
    cudaChk(hipGetDeviceCount(&deviceCount));

    // Check device exists
    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(0);
    }

    // Get handle for device 0 -> cuDevice
    cudaChk(hipDeviceGet(&cuDevice, 0));

    // Create context -> cuContext
    cudaChk(hipCtxCreate(&cuContext, 0, cuDevice));

    // Tests
    printf("matrix_add tests:");
    matrix_add_test();
    // printf('\n')
    // printf('dot_product tests:');
    // dot_product_test();
    // printf('\n');
    // printf('blas2 tests:');
    // blas2_test();

    return 0;
}