#include <Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include "helper.h"

#define cudaChk(err)  __cudaChk(err, __FILE__, __LINE__)

inline void __cudaChk(hipError_t err, const char* file, const int line)
{
    if (hipSuccess != err) {
        fprintf(stderr,
            "CUDA Driver API error = %04d from file <%s>, line %i.\n",
            err, file, line);
        exit(0);
    }
}

#define TIMER_INIT \
    LARGE_INTEGER frequency; \
    LARGE_INTEGER t1,t2; \
    double elapsedTime; \
    QueryPerformanceFrequency(&frequency);

#define TIMER_START QueryPerformanceCounter(&t1);

#define TIMER_STOP \
    QueryPerformanceCounter(&t2); \
    elapsedTime=(float)(t2.QuadPart-t1.QuadPart)/frequency.QuadPart; \
    printf("0.000000%f sec\n", elapsedTime);

void matrix_add_test() {

    TIMER_INIT
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    char* ModuleFile = "matrix_add.ptx";
    char* KernelName = "matrix_add";
    
    const int N = 1024;
    const int M = 1024;
    const int THREADS_PER_BLOCK = 1024;
    const int NUMBER_OF_BLOCKS = 32;
    const int GRID_DIM_X = 4;
    const int GRID_DIM_Y = 1;
    const int GRID_DIM_Z = 1;
    const int BLOCK_DIM_X = THREADS_PER_BLOCK;
    const int BLOCK_DIM_Y = 1;
    const int BLOCK_DIM_Z = 1;

    // Number of bytes to allocate for MxN matrix with type float
    int size = (M*N) * sizeof(float);

    // Load precompiled PTX from nvcc -> cuModule
    cudaChk(hipModuleLoad(&cuModule, ModuleFile));
        
    // Get function handle from module -> cuFunction
    cudaChk(hipModuleGetFunction(&cuFunction, cuModule, KernelName));

    // Allocate vectors in host memory
    float** a = (float**)malloc(size);
    float** b = (float**)malloc(size);
    float** c = (float**)malloc(size);

    // Allocate vectors in device memory
    hipDeviceptr_t dev_a, dev_b, dev_c;
    cudaChk(hipMalloc(&dev_a, size));
    cudaChk(hipMalloc(&dev_b, size));
    cudaChk(hipMalloc(&dev_c, size));

    // Initialize host vectors
    initMat(3, a, M, N);
    initMat(4, b, M, N);
    initMat(0, c, M, N);

    // Copy vectors from host memory to device memory
    cudaChk(hipMemcpyHtoD(dev_a, a, size));
    cudaChk(hipMemcpyHtoD(dev_b, b, size));

    // setup kernel arguments (using the simple kernel argument format)
    unsigned int sharedMemBytes = 1;
    hipStream_t hStream = 0;
    void* args[] = { &dev_a, &dev_b, &dev_c, M, N };

    // Launch the kernel on device
    TIMER_START
    cudaChk(hipModuleLaunchKernel(cuFunction, \
        GRID_DIM_X, GRID_DIM_Y, GRID_DIM_Z, \
        BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, \
        sharedMemBytes, hStream, args, 0));
    cudaChk(hipCtxSynchronize());
    TIMER_STOP

    // Check for errors
    hipError_t cuErrSync  = hipGetLastError();
    if (cuErrSync != hipSuccess) printf("sync error: %s\n", hipGetErrorString(cuErrSync));
    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess) printf("asyc error:  %s\n", hipGetErrorString(cuErrAsync));

    // Retrieve results from device & verify/use
    cudaChk(hipMemcpyDtoH(c, dev_c, size));

    // Check data for correctness
    checkElementsMat(7, c, M, N);

    // Free Host Memory
    free( a ); 
    free( b ); 
    free( c );

    // Free Device Memory
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
}
