#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "kernel.cuh"

__global__ void vecAdd_gridstride(float *result, float *a, float *b, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        result[i] = a[i] + b[i];
    }
}

void initWidth(float num, float* a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = num;
    }
}

void checkElementsAre(float target, float* vector, int N) {
    for (int i = 0; i < N; i++) {
        if (vector[i] != target) {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
        }
    }
    printf("SUCCESS! All values calculated correctly.\n");
}

// Error Handling Macro for CUDA Runtime
inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

int main() {
    const int N = 2 << 24;
    size_t size = N * sizeof(float);

    float* a;
    float* b;
    float* c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    initWidth(3, a, N);
    initWidth(4, b, N);
    initWidth(0, c, N);

    size_t numberOfBlocks;
    size_t threadsPerBlock;

    numberOfBlocks = 32;
    threadsPerBlock = 1024;

    hipError_t addVectorsErr;
    hipError_t asyncErr;
    
    printf("EXEC CONFIG: Number blocks: %d, number threads: %d\n", numberOfBlocks, threadsPerBlock);
    vecadd_1d<<<numberOfBlocks, threadsPerBlock>>>(c,a,b,N);

    hipError_t cuErrSync  = hipGetLastError();
    if (cuErrSync != hipSuccess) printf("sync error: %s\n", hipGetErrorString(cuErrSync));

    hipError_t cuErrAsync = hipDeviceSynchronize();
    if (cuErrAsync != hipSuccess) printf("asyc error:  %s\n", hipGetErrorString(asyncErr0));

    checkElementsAre(7, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}