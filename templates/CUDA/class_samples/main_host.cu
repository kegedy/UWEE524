#include <stdio.h>
#include <stdlib.h>
#include "kernel.cuh"

/*
    // STEP 0: Always check and handle errors (IMPLIED step...)
    // STEP 1: Define the platform ( = obtain the CUDA device and context)
    // STEP 2: Create and build the Module and Function
    // STEP 3: Setup memory objects to manage the input-output host and device data
    // STEP 4: Configure the kernel for execution - set up arguments, grid/index hierarchy
    // STEP 5: Launch the kernel
    // STEP 6: Retrieve results from device & verify/use

int main( ) {
    // Define static variables
    // Initialize GPU host API
    // Query for device information
    // Setup GPU host API environment and device program(s)
    // Allocate host memory variables h_
    // Initialize host memory vars
    // Allocate device memory vars d_
    // Set up kernel arguments on device
    // Copy host memory to device memory
    // Determine GPU device kernel execution configuration
    // Launch kernel on device
    // Wait for kernel execution to complete, check for errors
    // Retrieve results from device
    // Check data for correctness
    // Free Host Memory
    // Free Device Memory
}
*/

int main(int argc, char** argv)
{ 
    // Initialize
    cudaChk(hipInit(0));
    
    // Get number of devices supporting CUDA
    int deviceCount = 0;
    cudaChk(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(0);
    }

    // Get handle for device 0
    hipDevice_t cuDevice;
    cudaChk(hipDeviceGet(&cuDevice, 0));

    // Create context
    hipCtx_t cuContext;
    cudaChk(hipCtxCreate(&cuContext, 0, cuDevice));

    // Create module from binary file
    hipModule_t cuModule;

    // precompiled PTX or CUBIN from nvcc
    cudaChk(hipModuleLoad(&cuModule, "vecAdd_01.ptx"));
    
    // Get function handle from module; Note mangled name
    hipFunction_t vecAdd;
    cudaChk(hipModuleGetFunction(&vecAdd, cuModule, "_Z9vecAdd_01PfS_S_i"));

    // Allocate input vectors h_A etc. in host memory
    float* h_A = (float*)malloc(size);

    // Allocate vectors in device memory
    hipDeviceptr_t d_A;
    cudaChk(hipMalloc(&d_A, size));

    // Copy vectors from host memory to device memory
    cudaChk(hipMemcpyHtoD(d_A, h_A, size));

    // Invoke kernel
    int threadsPerBlock = 4;
    int blocksPerGrid = 4;

    // setup kernel arguments (using the simple kernel argument format)
    void* args[] = { &d_C, &d_A, &d_B, &N };
    
    cudaChk(hipModuleLaunchKernel(vecAdd, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0, 0, args, 0));
    cudaChk(hipCtxSynchronize());

    cudaChk(hipMemcpyDtoH(h_C, d_C, size));

    // Do data result verification routine...

    return 0;
}